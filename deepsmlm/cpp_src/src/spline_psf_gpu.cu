#include "hip/hip_runtime.h"
//
//  Created by Lucas Müller on 12.02.2020
//  Copyright © 2020 Lucas-Raphael Müller. All rights reserved.
//
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "spline_psf_gpu.cuh"

__device__
void kernel_computeDelta3D(spline *sp, 
float* delta_f, float* delta_dxf, float* delta_dyf, float* delta_dzf, 
float x_delta, float y_delta, float z_delta);

__global__
void fAt3Dj(spline *sp, float* rois, int roi_ix, int npx, int npy, 
int xc, int yc, int zc, float phot, float x_delta, float y_delta, float z_delta);

__global__
void fPSF(spline *sp, float *rois, int npx, int npy, 
float* xc_, float* yc_, float* zc_, float* phot_);

spline* d_spline_init(int xsize, int ysize, int zsize, const float *h_coeff) {
    

    spline* sp;
    sp = (spline *)malloc(sizeof(spline));

    sp->xsize = xsize;
    sp->ysize = ysize;
    sp->zsize = zsize;

    sp->roi_out_eps = 1e-10;
    sp->roi_out_deriv_eps = 0.0;

    sp->NV_PSP = 5;  
    sp->n_coeff = 64;

    int tsize = xsize * ysize * zsize * 64;

    float *d_coeff;
    hipMalloc(&d_coeff, tsize * sizeof(float));
    hipMemcpy(d_coeff, h_coeff, tsize * sizeof(float), hipMemcpyHostToDevice);

    sp->coeff = d_coeff;

    spline* d_sp;
    hipMalloc(&d_sp, sizeof(spline));
    hipMemcpy(d_sp, sp, sizeof(spline), hipMemcpyHostToDevice);

    return d_sp;
}

// Just a dummy for checking correct parsing from python
__global__
void check_spline(spline *d_sp) {
    printf("Checking spline ...\n");
    printf("\txs, ys, zs: %i %i %i\n", d_sp->xsize, d_sp->ysize, d_sp->zsize);

    printf("\tcoeff: ");
    for (int i = 0; i < 10; i++) {
        printf(" %2f", d_sp->coeff[i]);
    }
    printf("\n");
}

__device__
void kernel_computeDelta3D(spline *sp, 
    float* delta_f, float* delta_dxf, float* delta_dyf, float* delta_dzf, 
    float x_delta, float y_delta, float z_delta) {

    int i,j,k;
    float cx,cy,cz;

    cz = 1.0;
    for(i=0;i<4;i++){
        cy = 1.0;
        for(j=0;j<4;j++){
            cx = 1.0;
            for(k=0;k<4;k++){
                delta_f[i*16+j*4+k] = cz * cy * cx;
                if(k<3){
					delta_dxf[i*16+j*4+k+1] = ((float)k+1) * cz * cy * cx;
				}
				if(j<3){
					delta_dyf[i*16+(j+1)*4+k] = ((float)j+1) * cz * cy * cx;
				}
				if(i<3){
					delta_dzf[(i+1)*16+j*4+k] = ((float)i+1) * cz * cy * cx;
				}
                cx = cx * x_delta;
            }
            cy = cy * y_delta;
        }
        cz= cz * z_delta;
    }
}

__global__
void fAt3Dj(spline *sp, float* rois, int roi_ix, int npx, int npy,
    int xc, int yc, int zc, float phot, float x_delta, float y_delta, float z_delta) {
    
    int i = (blockIdx.x * blockDim.x + threadIdx.x) / npx;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) % npx;

     // allocate space for df, dxf, dyf, dzf
    __shared__ float delta_f[64], dxf[64], dyf[64], dzf[64];
    if (i == 0 and j == 0) {
        kernel_computeDelta3D(sp, delta_f, dxf, dyf, dzf, x_delta, y_delta, z_delta);
    }
    __syncthreads();

    xc += i;
    yc += j;
    
    float fv = 0;
    // Throw 0 for outside points (only x,y considered).
    if ((xc < 0) || (xc > sp->xsize-1) || (yc < 0) || (yc > sp->ysize-1)) {
        rois[roi_ix * npx * npy + i * npy + j] = sp->roi_out_eps;
        return;
    }

    xc = max(xc,0);
    xc = min(xc,sp->xsize-1);

    yc = max(yc,0);
    yc = min(yc,sp->ysize-1);

    zc = max(zc,0);
    zc = min(zc,sp->zsize-1);

    for (int i=0; i < 64; i++) {
        fv += delta_f[i] * sp->coeff[i * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
    }

    // write to global roi stack
    rois[roi_ix * npx * npy + i * npy + j] = phot * fv;
}

__global__
void fPSF(spline *sp, float *rois, int npx, int npy, float* xc_, float* yc_, float* zc_, float* phot_) {
    
    int r = blockIdx.x;  // roi index

    int x0, y0, z0;
    float xc, yc, zc, phot;
    float x_delta,y_delta,z_delta;

    xc = xc_[r];
    yc = yc_[r];
    zc = zc_[r];
    phot = phot_[r];

    /* Compute delta. Will be the same for all following px */
    x0 = (int)floor(xc);
    x_delta = xc - x0;

    y0 = (int)floor(yc);
    y_delta = yc - y0;

    z0 = (int)floor(zc);
    z_delta = zc - z0;

    fAt3Dj<<<1, npx * npy>>>(sp, rois, r, npx, npy, x0, y0, z0, phot, x_delta, y_delta, z_delta);
    hipDeviceSynchronize();
}

auto compute_rois(spline *d_sp, 
    const int n, const float *h_x, const float *h_y, const float *h_z, const float *h_phot) -> float* {

    // init cuda_err
    hipError_t err = hipSuccess;

    // setup n random localisations and ship them to GPU
    int roi_size_x = 13;
    int roi_size_y = 13;

    float *d_x, *d_y, *d_z, *d_phot;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_z, n * sizeof(float));
    hipMalloc(&d_phot, n * sizeof(float));
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_phot, h_phot, n * sizeof(float), hipMemcpyHostToDevice);

    // add output rois on host and device; 
    float* d_rois;
    hipMalloc(&d_rois, n * roi_size_x * roi_size_y * sizeof(float));
    hipMemset(d_rois, 0, n * roi_size_x * roi_size_y * sizeof(float));

    #if DEBUG
        check_spline<<<1,1>>>(d_sp);
        hipDeviceSynchronize();
    #endif

    // start n blocks which itself start number of px childs
    fPSF<<<n, 1>>>(d_sp, d_rois, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot);
    hipDeviceSynchronize();

    #if DEBUG
        std::cout << "Success.\n";
    #endif

    hipFree(&d_x);
    hipFree(&d_y);
    hipFree(&d_z);
    hipFree(&d_phot);

    return d_rois;  
}

// Wrapper around compute_roi function to put the results back to host
// 
auto compute_rois_h(spline *d_sp, const int n, const float *h_x, const float *h_y, const float *h_z, const float *h_phot) -> void {

    int roi_size_x = 13;
    int roi_size_y = 13;
    auto d_rois = compute_rois(d_sp, n, h_x, h_y, h_z, h_phot);
    
    // put results to host
    std::vector<float> h_rois(n * roi_size_x * roi_size_y);  // host
    hipMemcpy(h_rois.data(), d_rois, n * roi_size_x * roi_size_y * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(&d_rois);

    return;
}
